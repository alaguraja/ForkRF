#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
// Kernel definition
 __global__  void    MatAdd(float A, float B, float C)
 { 
 int i = threadIdx.x; 
 int j = threadIdx.y; 
 C= A + B;
 }
 void callCuda(int numBlocks,int b1,int b2){
  float A=5,B=10,C;
 dim3 threadsPerBlock(10, 10);
  MatAdd<<<numBlocks, threadsPerBlock>>>(&A,&B,&C);
 }
 int main() { 
 // Kernel invocation with one block of N * N * 1 threads
 int numBlocks = 1; 
 int i=0,j=0;
callCuda(1,10,10);

 printf("%f",C);
 return 1;
 }